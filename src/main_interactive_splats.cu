#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/iterator/discard_iterator.h>

#include "camera.hpp"
#include "gaussian.hpp"
#include "render_utils.hpp"
#include "cuda_helpers.hpp"
#include "camera_gpu.cu"
#include "ply_utils.cu"
#include "sdl_utils.cu"

const short WINDOW_WIDTH = 512*1.5;
const short WINDOW_HEIGHT = 512*1.5;

int main() {
    SDLApp app(WINDOW_WIDTH, WINDOW_HEIGHT);
    if (!app.init("Gaussian Splats Viewer")) {
        // Handle error (already printed in init)
        return 1;
    }

    // Load PLY file
    std::vector<float3> originalVertices;
    std::vector<float3> originalColors;
    int vertexCount = 0;
    std::vector<Gaussian3D> h_splats(0); // Will be resized in read_init_ply
    read_init_ply(originalVertices, vertexCount, originalColors, h_splats);
    

    // Calculate bounding sphere once
    BoundingSphere boundingSphere = calculateBoundingSphere(originalVertices);
    // Set up fixed camera parameters based on bounding sphere
    OrthoCameraParams camera;
    camera.imageWidth = WINDOW_WIDTH;
    camera.imageHeight = WINDOW_HEIGHT;

    
    // Calculate fixed camera bounds that will work for any rotation
    float aspectRatio = float(camera.imageWidth) / float(camera.imageHeight);
    float margin = 1.0f; // Add some margin around the object
    float radius = boundingSphere.radius * margin;

    // Calculate the half-width and half-height of our view volume
    float halfWidth = radius;
    float halfHeight = radius;
    if (aspectRatio > 1.0f) {
        halfWidth *= aspectRatio;
    } else {
        halfHeight /= aspectRatio;
    }
    
    // Center the camera bounds around the object's center
    camera.xMin = boundingSphere.center.x - halfWidth;
    camera.xMax = boundingSphere.center.x + halfWidth;
    camera.yMin = boundingSphere.center.y - halfHeight;
    camera.yMax = boundingSphere.center.y + halfHeight;


    // Allocate CUDA resources
    float4* d_image = nullptr;
    Gaussian3D* d_splats = nullptr;
    ProjectedSplat* d_outSplats = nullptr;
    float3* d_vertices = nullptr;
    
    hipMalloc(&d_image, camera.imageWidth * camera.imageHeight * sizeof(float4));
    hipMalloc(&d_splats, vertexCount * sizeof(Gaussian3D));
    hipMalloc(&d_outSplats, vertexCount * sizeof(ProjectedSplat));
    hipMalloc(&d_vertices, vertexCount * sizeof(float3));
    hipMemcpy(d_vertices, originalVertices.data(), vertexCount * sizeof(float3), hipMemcpyHostToDevice);
    float3* d_originalVertices = nullptr;  // Add this with other GPU allocations
    hipMalloc(&d_originalVertices, vertexCount * sizeof(float3));
    hipMemcpy(d_originalVertices, originalVertices.data(), vertexCount * sizeof(float3), hipMemcpyHostToDevice);

    int tileSize = 8;
    int tilesInX = camera.imageWidth / tileSize;
    int tilesInY = camera.imageHeight / tileSize;
    int totalTiles = tilesInX * tilesInY;
    
    int *d_tileRangeStart = nullptr, *d_tileRangeEnd = nullptr;
    hipMalloc(&d_tileRangeStart, totalTiles * sizeof(int));
    hipMalloc(&d_tileRangeEnd, totalTiles * sizeof(int));

    MouseState mouseState;
    bool running = true;
    SDL_Event event;

    FPSCounter fpsCounter;


    // Copy updated data to GPU
    hipMemcpy(d_splats, h_splats.data(), vertexCount * sizeof(Gaussian3D), hipMemcpyHostToDevice);

    while (running) {
        app.processEvents(mouseState, running, event);

        // We can set d_inVertices and d_outVertices to the same pointer because each thread 
        // maps to a different vertex in the array
        rotateVerticesOnGPU(
            d_originalVertices, d_vertices, vertexCount, d_splats,
            mouseState.totalRotationX, mouseState.totalRotationY, boundingSphere.center
        );

        // TODO: logically veryfiy we need to sync
        hipDeviceSynchronize();
        hipMemset(d_image, 0, camera.imageWidth * camera.imageHeight * sizeof(float4));  

        // Project Gaussians
        int blockSize = tileSize*tileSize;
        int gridSize = (vertexCount + blockSize - 1) / blockSize;
        projectGaussiansKernel<<<gridSize, blockSize>>>(
            d_splats, d_outSplats, vertexCount, camera, tileSize
        );
        hipDeviceSynchronize();

        // Sort by tileID then depth
        thrust::device_vector<unsigned long long> d_keys(vertexCount);
        thrust::transform(
            thrust::device_pointer_cast(d_outSplats),
            thrust::device_pointer_cast(d_outSplats + vertexCount),
            d_keys.begin(),
            [] __device__ (const ProjectedSplat& s) {
                return packTileDepth(s.tileID, s.depth);
            }
        );

        thrust::device_ptr<ProjectedSplat> d_splats_ptr(d_outSplats);
        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_splats_ptr);


        // 1) Fill tileRangeStart and tileRangeEnd on the device with -1
        thrust::device_ptr<int> d_startPtr(d_tileRangeStart);
        thrust::device_ptr<int> d_endPtr(d_tileRangeEnd);
        thrust::fill(d_startPtr, d_startPtr + totalTiles, -1);
        thrust::fill(d_endPtr,   d_endPtr + totalTiles,   -1);

        // 2) Create an array of indices [0, 1, 2, ...] for the splats
        thrust::device_vector<int> d_indices(vertexCount);
        thrust::sequence(d_indices.begin(), d_indices.end());

        // 3) Extract tileIDs from the sorted splats
        thrust::device_vector<int> d_tileIDs(vertexCount);
        thrust::transform(
            thrust::device_pointer_cast(d_outSplats),
            thrust::device_pointer_cast(d_outSplats + vertexCount),
            d_tileIDs.begin(),
            [] __device__ (const ProjectedSplat &s) {
                return s.tileID;
            }
        );

        // 4) reduce_by_key for min and max indices
        thrust::device_vector<int> d_tileIDsOut(vertexCount);
        thrust::device_vector<int> d_tileStartsOut(vertexCount);
        thrust::device_vector<int> d_tileEndsOut(vertexCount);

        // (a) find the FIRST index for each tile
        auto min_end = thrust::reduce_by_key(
            d_tileIDs.begin(), d_tileIDs.end(),  // keys
            d_indices.begin(),                   // values
            d_tileIDsOut.begin(),                // output keys
            d_tileStartsOut.begin(),             // output values (min indices)
            thrust::equal_to<int>(),
            thrust::minimum<int>()
        );

        // (b) find the LAST index for each tile
        auto max_end = thrust::reduce_by_key(
            d_tileIDs.begin(), d_tileIDs.end(),
            d_indices.begin(),
            thrust::make_discard_iterator(),    // we don't need to store keys again
            d_tileEndsOut.begin(),
            thrust::equal_to<int>(),
            thrust::maximum<int>()
        );

        // how many unique tiles did we actually get?
        int numUniqueTiles = static_cast<int>(min_end.first - d_tileIDsOut.begin());

        // 5) Scatter results directly on the GPU
        // We'll launch a kernel to write tileRangeStart[tile], tileRangeEnd[tile].
        {
            int blockSize = tileSize*tileSize;
            int gridSize = (numUniqueTiles + blockSize - 1) / blockSize;
            scatterTileRanges<<<gridSize, blockSize>>>(
                thrust::raw_pointer_cast(d_tileIDsOut.data()),
                thrust::raw_pointer_cast(d_tileStartsOut.data()),
                thrust::raw_pointer_cast(d_tileEndsOut.data()),
                d_tileRangeStart,
                d_tileRangeEnd,
                numUniqueTiles,
                totalTiles
            );
            hipDeviceSynchronize();
        }

        // Render
        dim3 blocks(totalTiles, 1, 1);
        dim3 threads(tileSize * tileSize, 1, 1);
        tiledBlendingKernel<<<blocks, threads>>>(
            d_outSplats, d_image, d_tileRangeStart, d_tileRangeEnd,
            camera, tileSize
        );
        hipDeviceSynchronize();

        // Copy result back and update texture
        std::vector<float4> h_image(camera.imageWidth * camera.imageHeight);
        hipMemcpy(h_image.data(), d_image, camera.imageWidth * camera.imageHeight * sizeof(float4), hipMemcpyDeviceToHost);

        std::vector<Uint32> pixels(camera.imageWidth * camera.imageHeight);
        for (int i = 0; i < camera.imageWidth * camera.imageHeight; i++) {
            float4 px = h_image[i];
            Uint8 r = (Uint8)(255.f * fminf(fmaxf(px.x, 0.f), 1.f));
            Uint8 g = (Uint8)(255.f * fminf(fmaxf(px.y, 0.f), 1.f));
            Uint8 b = (Uint8)(255.f * fminf(fmaxf(px.z, 0.f), 1.f));
            Uint8 a = (Uint8)(255.f * fminf(fmaxf(px.w, 0.f), 1.f));
            //Creates RGBA ordering in memory, but SDL interprets this as ABGR (reading right-to-left)
            //pixels[i] = (r << 24) | (g << 16) | (b << 8) | a;
            //- Creates ABGR ordering in memory, SDL correctly interprets as RGBA
            pixels[i] = (r << 0) | (g << 8) | (b << 16) | (a << 24); 
        }

        app.renderFrame(pixels, fpsCounter, camera);
    }

    // Cleanup
    hipFree(d_splats);
    hipFree(d_outSplats);
    hipFree(d_vertices);
    hipFree(d_image);
    hipFree(d_tileRangeStart);
    hipFree(d_tileRangeEnd);
    hipFree(d_originalVertices);

    return 0;
}