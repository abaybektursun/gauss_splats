#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "camera.hpp"
#include "gaussian.hpp"
#include "render_utils.hpp"
#include "cuda_helpers.hpp"
#include "camera_gpu.cu"
#include "ply_utils.cu"
#include "sdl_utils.cu"

const short WINDOW_WIDTH = 512*1.5;
const short WINDOW_HEIGHT = 512*1.5;

int main() {
    SDLApp app(WINDOW_WIDTH, WINDOW_HEIGHT);
    if (!app.init("Gaussian Splats Viewer")) {
        return 1;
    }

    // Load PLY file
    std::vector<float3> originalVertices;
    std::vector<float3> originalColors;
    int vertexCount = 0;
    std::vector<Gaussian3D> h_splats(0); // Will be resized in read_init_ply
    read_init_ply(originalVertices, vertexCount, originalColors, h_splats);
    
    BoundingSphere boundingSphere = calculateBoundingSphere(originalVertices);
    OrthoCameraParams camera;
    camera.imageWidth = WINDOW_WIDTH;
    camera.imageHeight = WINDOW_HEIGHT;

    
    // Calculate fixed camera bounds that will work for any rotation
    float aspectRatio = float(camera.imageWidth) / float(camera.imageHeight);
    float margin = 1.0f; // >1.0 to add some margin around the object
    float radius = boundingSphere.radius * margin;

    // Calculate the half-width and half-height of our view volume
    float halfWidth = radius;
    float halfHeight = radius;
    if (aspectRatio > 1.0f) {
        halfWidth *= aspectRatio;
    } else {
        halfHeight /= aspectRatio;
    }
    
    // Center the camera bounds around the object's center
    camera.xMin = boundingSphere.center.x - halfWidth;
    camera.xMax = boundingSphere.center.x + halfWidth;
    camera.yMin = boundingSphere.center.y - halfHeight;
    camera.yMax = boundingSphere.center.y + halfHeight;


    // Allocate CUDA resources
    GPUData gpuData;

    const int tileSize = 8;
    const int tilesInX = camera.imageWidth / tileSize;
    const int tilesInY = camera.imageHeight / tileSize;
    const int totalTiles = tilesInX * tilesInY;
    const int blockSize = tileSize*tileSize;
    const int gridSize = (vertexCount + blockSize - 1) / blockSize;
    
    hipMalloc(&gpuData.d_tileRangeStart, totalTiles * sizeof(int));
    hipMalloc(&gpuData.d_tileRangeEnd, totalTiles * sizeof(int));
    hipMalloc(&gpuData.d_image, camera.imageWidth * camera.imageHeight * sizeof(float4));
    hipMalloc(&gpuData.d_outSplats, vertexCount * sizeof(ProjectedSplat));
    hipMalloc(&gpuData.d_vertices, vertexCount * sizeof(float3));
    hipMemcpy(gpuData.d_vertices, originalVertices.data(), vertexCount * sizeof(float3), hipMemcpyHostToDevice);

    hipMalloc(&gpuData.d_originalVertices, vertexCount * sizeof(float3));
    hipMemcpy(gpuData.d_originalVertices, originalVertices.data(), vertexCount * sizeof(float3), hipMemcpyHostToDevice);
    
    hipMalloc(&gpuData.d_splats, vertexCount * sizeof(Gaussian3D));
    hipMemcpy(gpuData.d_splats, h_splats.data(), vertexCount * sizeof(Gaussian3D), hipMemcpyHostToDevice);


    SDL_Event event;
    bool running = true;
    MouseState mouseState;
    while (running) {
        app.processEvents(mouseState, running, event);

        // We can set d_inVertices and d_outVertices to the same pointer because each thread 
        // maps to a different vertex in the array
        rotateVerticesOnGPU(
            gpuData.d_originalVertices, gpuData.d_vertices, vertexCount, gpuData.d_splats,
            mouseState.totalRotationX, mouseState.totalRotationY, boundingSphere.center
        );

        // Clear image
        hipMemset(gpuData.d_image, 0, camera.imageWidth * camera.imageHeight * sizeof(float4));  

        // Project Gaussians
        projectGaussiansKernel<<<gridSize, blockSize>>>(
            gpuData.d_splats, gpuData.d_outSplats, vertexCount, camera, tileSize
        );
        hipDeviceSynchronize();

        // Sort by tileID then depth
        sortSplats(gpuData, vertexCount);

        generateTileRanges(gpuData.d_outSplats, totalTiles, tileSize, vertexCount, gpuData.d_tileRangeStart, gpuData.d_tileRangeEnd);

        // Render
        dim3 blocks(totalTiles, 1, 1);
        dim3 threads(tileSize * tileSize, 1, 1);
        tiledBlendingKernel<<<blocks, threads>>>(
            gpuData.d_outSplats, gpuData.d_image, gpuData.d_tileRangeStart, gpuData.d_tileRangeEnd,
            camera, tileSize
        );
        hipDeviceSynchronize();

        // Copy result back and update texture
        std::vector<float4> h_image(camera.imageWidth * camera.imageHeight);
        hipMemcpy(h_image.data(), gpuData.d_image, camera.imageWidth * camera.imageHeight * sizeof(float4), hipMemcpyDeviceToHost);

        std::vector<Uint32> pixels(camera.imageWidth * camera.imageHeight);
        for (int i = 0; i < camera.imageWidth * camera.imageHeight; i++) {
            float4 px = h_image[i];
            Uint8 r = (Uint8)(255.f * fminf(fmaxf(px.x, 0.f), 1.f));
            Uint8 g = (Uint8)(255.f * fminf(fmaxf(px.y, 0.f), 1.f));
            Uint8 b = (Uint8)(255.f * fminf(fmaxf(px.z, 0.f), 1.f));
            Uint8 a = (Uint8)(255.f * fminf(fmaxf(px.w, 0.f), 1.f));
            //Creates RGBA ordering in memory, but SDL interprets this as ABGR (reading right-to-left)
            //pixels[i] = (r << 24) | (g << 16) | (b << 8) | a;
            //- Creates ABGR ordering in memory, SDL correctly interprets as RGBA
            pixels[i] = (r << 0) | (g << 8) | (b << 16) | (a << 24); 
        }

        app.renderFrame(pixels, camera);
    }

    releaseGPUData(gpuData);

    return 0;
}