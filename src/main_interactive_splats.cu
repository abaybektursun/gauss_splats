#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <random>
#include <vector>
#include <SDL2/SDL.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>

#include "camera.hpp"
#include "gaussian.hpp"
#include "render_utils.hpp"
#include "cuda_helpers.hpp"

#include "tinyply.h"

const int WINDOW_WIDTH = 512*2;
const int WINDOW_HEIGHT = 512*2;


struct BoundingSphere {
    float3 center;
    float radius;
};

struct MouseState {
    bool leftButtonDown = false;
    int lastX = 0;
    int lastY = 0;
    float rotationX = 0.0f;
    float rotationY = 0.0f;
};

BoundingSphere calculateBoundingSphere(const std::vector<float3>& vertices) {
    BoundingSphere sphere;
    
    // Calculate center as average of all vertices
    sphere.center = make_float3(0.0f, 0.0f, 0.0f);
    for (const auto& v : vertices) {
        sphere.center.x += v.x;
        sphere.center.y += v.y;
        sphere.center.z += v.z;
    }
    sphere.center.x /= vertices.size();
    sphere.center.y /= vertices.size();
    sphere.center.z /= vertices.size();
    
    // Calculate radius as maximum distance from center to any vertex
    sphere.radius = 0.0f;
    for (const auto& v : vertices) {
        float dx = v.x - sphere.center.x;
        float dy = v.y - sphere.center.y;
        float dz = v.z - sphere.center.z;
        float dist = sqrt(dx*dx + dy*dy + dz*dz);
        sphere.radius = std::max(sphere.radius, dist);
    }
    
    return sphere;
}

void handleMouseEvent(SDL_Event& event, MouseState& mouseState) {
    switch(event.type) {
        case SDL_MOUSEBUTTONDOWN:
            if (event.button.button == SDL_BUTTON_LEFT) {
                mouseState.leftButtonDown = true;
                mouseState.lastX = event.button.x;
                mouseState.lastY = event.button.y;
            }
            break;
            
        case SDL_MOUSEBUTTONUP:
            if (event.button.button == SDL_BUTTON_LEFT) {
                mouseState.leftButtonDown = false;
            }
            break;
            
        case SDL_MOUSEMOTION:
            if (mouseState.leftButtonDown) {
                int deltaX = event.motion.x - mouseState.lastX;
                int deltaY = event.motion.y - mouseState.lastY;
                
                mouseState.rotationX += deltaY * 0.005f;
                mouseState.rotationY += deltaX * 0.005f;
                
                mouseState.lastX = event.motion.x;
                mouseState.lastY = event.motion.y;
            }
            break;
    }
}

void rotateVertices(std::vector<float3>& vertices, const MouseState& mouseState, const float3& center) {
    // Create rotation matrix around center point
    glm::mat4 toOrigin = glm::translate(glm::mat4(1.0f), glm::vec3(-center.x, -center.y, -center.z));
    glm::mat4 fromOrigin = glm::translate(glm::mat4(1.0f), glm::vec3(center.x, center.y, center.z));
    
    glm::mat4 rotation = glm::rotate(glm::mat4(1.0f), mouseState.rotationY, glm::vec3(0.0f, 1.0f, 0.0f));
    rotation = glm::rotate(rotation, mouseState.rotationX, glm::vec3(1.0f, 0.0f, 0.0f));
    
    glm::mat4 transform = fromOrigin * rotation * toOrigin;
    
    for (auto& vertex : vertices) {
        glm::vec4 rotated = transform * glm::vec4(vertex.x, vertex.y, vertex.z, 1.0f);
        vertex.x = rotated.x;
        vertex.y = rotated.y;
        vertex.z = rotated.z;
    }
}

int main() {
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL initialization failed: " << SDL_GetError() << std::endl;
        return 1;
    }

    SDL_Window* window = SDL_CreateWindow(
        "Gaussian Splats Viewer",
        SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED,
        WINDOW_WIDTH, WINDOW_HEIGHT,
        SDL_WINDOW_SHOWN
    );


    if (!window) {
        std::cerr << "Window creation failed: " << SDL_GetError() << std::endl;
        SDL_Quit();
        return 1;
    }

    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    if (!renderer) {
        std::cerr << "Renderer creation failed: " << SDL_GetError() << std::endl;
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }

    // Replace texture creation with:
    SDL_Texture* texture = SDL_CreateTexture(
        renderer,
        SDL_PIXELFORMAT_RGBA32,
        SDL_TEXTUREACCESS_STREAMING,
        WINDOW_WIDTH, WINDOW_HEIGHT
    );

    if (!texture) {
        std::cerr << "Texture creation failed: " << SDL_GetError() << std::endl;
        SDL_DestroyRenderer(renderer);
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }

    // Load PLY file
    std::string file_path = "/workspaces/gauss_splats/airplane.ply.txt";
    std::vector<float3> originalVertices;
    std::vector<int> faces;
    std::vector<float3> originalColors;
    int vertexCount = 0, faceCount = 0;
    
    // OLD CODE
    /*if (!loadPlyFile(file_path, originalVertices, faces, vertexCount, faceCount)) {
        SDL_DestroyTexture(texture);
        SDL_DestroyRenderer(renderer);
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }*/
    // New code 
    try {
        // Open the PLY file
        std::ifstream file("/workspaces/gauss_splats/Tree.ply", std::ios::binary);
        if (!file) throw std::runtime_error("Failed to open PLY file.");

        tinyply::PlyFile plyFile;
        plyFile.parse_header(file);

        // Read vertex properties
        std::shared_ptr<tinyply::PlyData> vertices, colors;
        vertices = plyFile.request_properties_from_element("vertex", {"x", "y", "z"});
        colors = plyFile.request_properties_from_element("vertex", {"red", "green", "blue"});

        plyFile.read(file);

        // Process vertex data
        std::vector<float> vertexBuffer(vertices->count * 3);
        std::memcpy(vertexBuffer.data(), vertices->buffer.get(), vertices->buffer.size_bytes());

        std::vector<uint8_t> colorBuffer(colors->count * 3);
        std::memcpy(colorBuffer.data(), colors->buffer.get(), colors->buffer.size_bytes());

        std::cout << "Read " << vertices->count << " points with color data." << std::endl;

        // Convert and copy data to originalVertices
        originalVertices.resize(vertices->count);
        for (size_t i = 0; i < vertices->count; i++) {
            originalVertices[i] = make_float3(
                vertexBuffer[i * 3 + 0],
                vertexBuffer[i * 3 + 1],
                vertexBuffer[i * 3 + 2]
            );
        }
        vertexCount = vertices->count;

        // Same for colors
        originalColors.resize(vertices->count);
        for (size_t i = 0; i < vertices->count; i++) {
            originalColors[i] = make_float3(
                colorBuffer[i * 3 + 0] / 255.0f,
                colorBuffer[i * 3 + 1] / 255.0f,
                colorBuffer[i * 3 + 2] / 255.0f
            );
        }
        
    } catch (const std::exception &e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    // Calculate bounding sphere once
    BoundingSphere boundingSphere = calculateBoundingSphere(originalVertices);
    
    // Set up fixed camera parameters based on bounding sphere
    OrthoCameraParams camera;
    camera.imageWidth = WINDOW_WIDTH;
    camera.imageHeight = WINDOW_HEIGHT;
    
    // Calculate fixed camera bounds that will work for any rotation
    float aspectRatio = float(camera.imageWidth) / float(camera.imageHeight);
    float margin = 1.1f; // Add some margin around the object
    float radius = boundingSphere.radius * margin;

    // Calculate the half-width and half-height of our view volume
    float halfWidth = radius;
    float halfHeight = radius;
    if (aspectRatio > 1.0f) {
        halfWidth *= aspectRatio;
    } else {
        halfHeight /= aspectRatio;
    }
    
    // Center the camera bounds around the object's center
    camera.xMin = boundingSphere.center.x - halfWidth;
    camera.xMax = boundingSphere.center.x + halfWidth;
    camera.yMin = boundingSphere.center.y - halfHeight;
    camera.yMax = boundingSphere.center.y + halfHeight;


    // Allocate CUDA resources
    float4* d_image = nullptr;
    Gaussian3D* d_splats = nullptr;
    ProjectedSplat* d_outSplats = nullptr;
    float3* d_vertices = nullptr;
    
    hipMalloc(&d_image, camera.imageWidth * camera.imageHeight * sizeof(float4));
    hipMalloc(&d_splats, vertexCount * sizeof(Gaussian3D));
    hipMalloc(&d_outSplats, vertexCount * sizeof(ProjectedSplat));
    hipMalloc(&d_vertices, vertexCount * sizeof(float3));

    int tileSize = 16;
    int tilesInX = camera.imageWidth / tileSize;
    int tilesInY = camera.imageHeight / tileSize;
    int totalTiles = tilesInX * tilesInY;
    
    int *d_tileRangeStart = nullptr, *d_tileRangeEnd = nullptr;
    hipMalloc(&d_tileRangeStart, totalTiles * sizeof(int));
    hipMalloc(&d_tileRangeEnd, totalTiles * sizeof(int));

    MouseState mouseState;
    bool running = true;
    SDL_Event event;

    // TODO: remove
    int TMP_ITERS = 1000;
    while (running) {
        // TODO: remove
        if (TMP_ITERS-- <= 0) {
            break;
        }
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                running = false;
            }
            handleMouseEvent(event, mouseState);
        }

        // Create a copy of vertices for this frame and rotate them
        std::vector<float3> vertices = originalVertices;
        rotateVertices(vertices, mouseState, boundingSphere.center);

        // Update Gaussians with rotated positions
        std::vector<Gaussian3D> h_splats(vertexCount);
        for (int i = 0; i < vertexCount; i++) {
            h_splats[i].position = vertices[i];
            h_splats[i].scale = make_float3(0.005f, 0.005f, 0.005f);
            h_splats[i].opacity = 0.5f;
            // Read the colors from the original data
            h_splats[i].color = make_float3(
                originalColors[i].x,
                originalColors[i].y,
                originalColors[i].z
            );
            h_splats[i].intensity = 0.0f;
            h_splats[i].rotation = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
        }

        // Clear image and copy updated data to GPU
        hipMemset(d_image, 0, camera.imageWidth * camera.imageHeight * sizeof(float4));
        hipMemcpy(d_splats, h_splats.data(), vertexCount * sizeof(Gaussian3D), hipMemcpyHostToDevice);

        // Project Gaussians
        int blockSize = 256;
        int gridSize = (vertexCount + blockSize - 1) / blockSize;
        projectGaussiansKernel<<<gridSize, blockSize>>>(
            d_splats, d_outSplats, vertexCount, camera, 16
        );
        hipDeviceSynchronize();

        // Sort by tileID then depth
        thrust::device_vector<unsigned long long> d_keys(vertexCount);
        thrust::transform(
            thrust::device_pointer_cast(d_outSplats),
            thrust::device_pointer_cast(d_outSplats + vertexCount),
            d_keys.begin(),
            [] __device__ (const ProjectedSplat& s) {
                return packTileDepth(s.tileID, s.depth);
            }
        );

        thrust::device_ptr<ProjectedSplat> d_splats_ptr(d_outSplats);
        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_splats_ptr);

        // Compute and copy tile ranges
        std::vector<ProjectedSplat> h_splatsSorted(vertexCount);
        hipMemcpy(h_splatsSorted.data(), d_outSplats, vertexCount * sizeof(ProjectedSplat), hipMemcpyDeviceToHost);

        std::vector<int> h_tileRangeStart(totalTiles, -1);
        std::vector<int> h_tileRangeEnd(totalTiles, -1);
        computeTileRanges(h_splatsSorted, totalTiles, h_tileRangeStart, h_tileRangeEnd);

        hipMemcpy(d_tileRangeStart, h_tileRangeStart.data(), totalTiles * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_tileRangeEnd, h_tileRangeEnd.data(), totalTiles * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_outSplats, h_splatsSorted.data(), vertexCount * sizeof(ProjectedSplat), hipMemcpyHostToDevice);

        // Render
        dim3 blocks(totalTiles, 1, 1);
        dim3 threads(tileSize * tileSize, 1, 1);
        tiledBlendingKernel<<<blocks, threads>>>(
            d_outSplats, d_image, d_tileRangeStart, d_tileRangeEnd,
            camera, tileSize
        );
        hipDeviceSynchronize();

        // Copy result back and update texture
        std::vector<float4> h_image(camera.imageWidth * camera.imageHeight);
        hipMemcpy(h_image.data(), d_image, camera.imageWidth * camera.imageHeight * sizeof(float4), hipMemcpyDeviceToHost);

        std::vector<Uint32> pixels(camera.imageWidth * camera.imageHeight);
        for (int i = 0; i < camera.imageWidth * camera.imageHeight; i++) {
            float4 px = h_image[i];
            Uint8 r = (Uint8)(255.f * fminf(fmaxf(px.x, 0.f), 1.f));
            Uint8 g = (Uint8)(255.f * fminf(fmaxf(px.y, 0.f), 1.f));
            Uint8 b = (Uint8)(255.f * fminf(fmaxf(px.z, 0.f), 1.f));
            Uint8 a = (Uint8)(255.f * fminf(fmaxf(px.w, 0.f), 1.f));
            //Creates RGBA ordering in memory, but SDL interprets this as ABGR (reading right-to-left)
            //pixels[i] = (r << 24) | (g << 16) | (b << 8) | a;
            //- Creates ABGR ordering in memory, SDL correctly interprets as RGBA
            pixels[i] = (r << 0) | (g << 8) | (b << 16) | (a << 24); 
        }

        SDL_UpdateTexture(texture, NULL, pixels.data(), camera.imageWidth * sizeof(Uint32));
        SDL_RenderClear(renderer);
        SDL_RenderCopy(renderer, texture, NULL, NULL);
        SDL_RenderPresent(renderer);

        SDL_Delay(16); // roughly 60 FPS
    }

    // Cleanup
    hipFree(d_splats);
    hipFree(d_outSplats);
    hipFree(d_vertices);
    hipFree(d_image);
    hipFree(d_tileRangeStart);
    hipFree(d_tileRangeEnd);

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}