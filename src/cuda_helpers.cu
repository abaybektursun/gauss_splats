#include "cuda_helpers.hpp"


void releaseGPUData(GPUData& data) {
    hipFree(data.d_image);
    hipFree(data.d_splats);
    hipFree(data.d_outSplats);
    hipFree(data.d_vertices);
    hipFree(data.d_originalVertices);
    hipFree(data.d_tileRangeStart);
    hipFree(data.d_tileRangeEnd);
    hipFree(data.d_splatCounts);
    hipFree(data.d_splatOffsets);
    hipFree(data.d_tileSplats);
    std::cout << "\n";
}