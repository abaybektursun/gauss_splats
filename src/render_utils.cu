#include "hip/hip_runtime.h"
#include <cstdio>  // for printf (if needed)
#include "render_utils.hpp"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

/**
 * Device-side function to do orthographic projection.
 */
__device__ void orthographicProject(float x, float y,
                                    const OrthoCameraParams& cam,
                                    int& outU, int& outV)
{
    float normalizedX = (x - cam.xMin) / (cam.xMax - cam.xMin);
    float normalizedY = (y - cam.yMin) / (cam.yMax - cam.yMin);

    outU = static_cast<int>(normalizedX * (cam.imageWidth  - 1));
    outV = static_cast<int>(normalizedY * (cam.imageHeight - 1));
}

/**
 * Kernel: project 3D Gaussians to 2D splats.
 */
__global__
void projectGaussiansKernel(const Gaussian3D* d_gaussians,
                            ProjectedSplat* d_outSplats,
                            int numGaussians,
                            OrthoCameraParams cam,
                            int tile_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numGaussians) return;

    Gaussian3D g = d_gaussians[idx];

    float x = g.position.x;
    float y = g.position.y;
    float z = g.position.z;

    float s_x = g.scale.x;
    float s_y = g.scale.y;
    float s_z = g.scale.z;

    // Scale Matrix: S
    // | s_x  0   0 |
    // | 0   s_y  0 |
    // | 0    0  s_z|

    // Convert Quaternion to Rotation Matrix: R
    // | 1-2y^2-2z^2  2xy-2zw      2xz+2yw     |
    // | 2xy+2zw      1-2x^2-2z^2  2yz-2xw     |
    // | 2xz-2yw      2yz+2xw      1-2x^2-2y^2 |

    float  R11 = 1 - 2 * g.rotation.y * g.rotation.y - 2 * g.rotation.z * g.rotation.z;
    float  R12 = 2 * g.rotation.x * g.rotation.y - 2 * g.rotation.z * g.rotation.w;
    float  R13 = 2 * g.rotation.x * g.rotation.z + 2 * g.rotation.y * g.rotation.w;
    float  R21 = 2 * g.rotation.x * g.rotation.y + 2 * g.rotation.z * g.rotation.w;
    float  R22 = 1 - 2 * g.rotation.x * g.rotation.x - 2 * g.rotation.z * g.rotation.z;
    float  R23 = 2 * g.rotation.y * g.rotation.z - 2 * g.rotation.x * g.rotation.w;
    float  R31 = 2 * g.rotation.x * g.rotation.z - 2 * g.rotation.y * g.rotation.w;
    float  R32 = 2 * g.rotation.y * g.rotation.z + 2 * g.rotation.x * g.rotation.w;
    float  R33 = 1 - 2 * g.rotation.x * g.rotation.x - 2 * g.rotation.y * g.rotation.y;

    // Sigma = R * S * R^T = M * R^T; 
    // M = R * S
    float M11 = R11 * s_x;  float M12 = R12 * s_y; float M13 = R13 * s_z;
    float M21 = R21 * s_x;  float M22 = R22 * s_y; float M23 = R23 * s_z;
    float M31 = R31 * s_x;  float M32 = R32 * s_y; float M33 = R33 * s_z;

    // Sigma = M * R^T
    // R^T = | R11 R21 R31 |
    //       | R12 R22 R32 |
    //       | R13 R23 R33 |

    float sigma11 = M11 * R11 + M12 * R12 + M13 * R13;
    float sigma12 = M11 * R21 + M12 * R22 + M13 * R23;
    float sigma13 = M11 * R31 + M12 * R32 + M13 * R33;
    float sigma21 = M21 * R11 + M22 * R12 + M23 * R13;
    float sigma22 = M21 * R21 + M22 * R22 + M23 * R23;
    float sigma23 = M21 * R31 + M22 * R32 + M23 * R33;
    float sigma31 = M31 * R11 + M32 * R12 + M33 * R13;
    float sigma32 = M31 * R21 + M32 * R22 + M33 * R23;
    float sigma33 = M31 * R31 + M32 * R32 + M33 * R33;
    
    // Screen-space covariance matrix
    // | s_x^2*sigma11  s_x*s_y*sigma12 |
    // | s_x*s_y*sigma12 s_y^2*sigma22  |
    float screenSigma11 = s_x * s_x * sigma11; float screenSigma12 = s_x * s_y * sigma12;
    float screenSigma22 = s_y * s_y * sigma22; float screenSigma13 = s_x * s_y * sigma22;

    int u, v;
    orthographicProject(x, y, cam, u, v);

    if (u < 0 || u >= cam.imageWidth ||
        v < 0 || v >= cam.imageHeight)
    {
        d_outSplats[idx].tileID = -1;
        return;
    }

    // Determine tile
    int tileX  = u / tile_size;
    int tileY  = v / tile_size;
    int tileID = tileY * (cam.imageWidth / tile_size) + tileX;

    d_outSplats[idx].tileID  = tileID;
    d_outSplats[idx].depth   = z;
    d_outSplats[idx].pixelX  = u;
    d_outSplats[idx].pixelY  = v;
    // Pointer to the original Gaussian
    d_outSplats[idx].gaussian = (Gaussian3D*)&d_gaussians[idx];
}

/**
 * Device-side inline function for alpha blending.
 */
__device__ inline
void alphaBlend(float4& dest, const float4& src)
{
    float alphaAccum = dest.w;
    float alphaSplat = src.w;

    if (alphaAccum < 1.0f && alphaSplat > 0.0f) {
        float oneMinusA = 1.0f - alphaAccum;
        dest.x += oneMinusA * alphaSplat * src.x;
        dest.y += oneMinusA * alphaSplat * src.y;
        dest.z += oneMinusA * alphaSplat * src.z;
        dest.w += oneMinusA * alphaSplat;
    }
}

/**
 * Kernel: for each tile (block), blend all its splats in thread order.
 */
__global__
void tiledBlendingKernel(const ProjectedSplat* d_inSplats,
                         float4*               d_outImage,
                         const int*            d_tileRangeStart,
                         const int*            d_tileRangeEnd,
                         OrthoCameraParams     cam,
                         int                   tile_size)
{
    int tileIndex = blockIdx.x;
    int start = d_tileRangeStart[tileIndex];
    int end   = d_tileRangeEnd[tileIndex];
    if (start >= end) return;

    /* For 64x64 image and tile_size=16 (64^2/16^2=16):
    0    1   2   3     <- blockIdx.x TileY 0
    4    5   6   7     <- blockIdx.x TileY 1
    8    9  10  11     <- blockIdx.x TileY 2
    12  13  14  15     <- blockIdx.x TileY 3
    */

    int tilesInX = cam.imageWidth / tile_size;
    int tileX    = tileIndex % tilesInX;
    int tileY    = tileIndex / tilesInX;

    // Where does this tile start in the image?
    int tileOriginX = tileX * tile_size;
    int tileOriginY = tileY * tile_size;

    int localIdx = threadIdx.x;
    if (localIdx >= tile_size * tile_size) return;

    // Coordinates in the block
    int localY = localIdx / tile_size;
    int localX = localIdx % tile_size;
    // Coordinates in the image
    int globalX = tileOriginX + localX;
    int globalY = tileOriginY + localY;
    int globalPixelIdx = globalY * cam.imageWidth + globalX;

    // TODO: make this dynamic based on tile_size
    __shared__ float4 tilePixels[256];  // tile_size=16 => 16*16=256
    tilePixels[localIdx] = d_outImage[globalPixelIdx];
    __syncthreads();

    // Blend each splat in range
    for (int i = start; i < end; i++) {
        ProjectedSplat s = d_inSplats[i];
        Gaussian3D* gPtr = s.gaussian;

        // Build the source color
        float4 srcColor = make_float4(gPtr->color.x,
                                      gPtr->color.y,
                                      gPtr->color.z,
                                      gPtr->opacity);

        // === NEW: If globalX, globalY is within +/-1 of s.pixelX, s.pixelY
        //           then alphaBlend. That covers a 3x3 block for each splat.
        //           Increase or decrease this range as you like.
        const int radius = 1; // half-size of your dot
        if (abs(globalX - s.pixelX) <= radius &&
            abs(globalY - s.pixelY) <= radius)
        {
            // Now do your alphaBlend
            alphaBlend(tilePixels[localIdx], srcColor);

            // Optionally break if fully opaque
            if (tilePixels[localIdx].w > 0.999f) {
                break;
            }
        }
    }


    __syncthreads();
    d_outImage[globalPixelIdx] = tilePixels[localIdx];
}

/**
 * CPU utility: compute tileRangeStart / tileRangeEnd from sorted splats on the host.
 */
void computeTileRanges(std::vector<ProjectedSplat>& h_sortedSplats,
                       int totalTiles,
                       std::vector<int>& tileRangeStart,
                       std::vector<int>& tileRangeEnd)
{
    std::fill(tileRangeStart.begin(), tileRangeStart.end(), -1);
    std::fill(tileRangeEnd.begin(),   tileRangeEnd.end(),   -1);

    if (h_sortedSplats.empty()) return;

    int currentTile = h_sortedSplats[0].tileID;
    if (currentTile >= 0 && currentTile < totalTiles) {
        tileRangeStart[currentTile] = 0;
    }

    for (int i = 1; i < (int)h_sortedSplats.size(); i++) {
        int prevTile = h_sortedSplats[i-1].tileID;
        int thisTile = h_sortedSplats[i].tileID;
        if (thisTile != prevTile) {
            if (prevTile >= 0 && prevTile < totalTiles) {
                tileRangeEnd[prevTile] = i;
            }
            if (thisTile >= 0 && thisTile < totalTiles &&
                tileRangeStart[thisTile] == -1)
            {
                tileRangeStart[thisTile] = i;
            }
        }
    }

    int lastTile = h_sortedSplats.back().tileID;
    if (lastTile >= 0 && lastTile < totalTiles) {
        tileRangeEnd[lastTile] = (int)h_sortedSplats.size();
    }
}

void orbitCamera(float angleZ, OrthoCameraParams& camera, const float3& sceneMin, const float3& sceneMax)
{
    float cx = 0.5f * (sceneMin.x + sceneMax.x);
    float cy = 0.5f * (sceneMin.y + sceneMax.y);
    float dx = 0.5f * (sceneMax.x - sceneMin.x);
    float dy = 0.5f * (sceneMax.y - sceneMin.y);

    float cosA = cosf(angleZ);
    float sinA = sinf(angleZ);

    // Define the four corners of the bounding box relative to the center
    float corners[4][2] = {
        {-dx, -dy},
        { dx, -dy},
        { dx,  dy},
        {-dx,  dy}
    };

    float minX = 1e30f, maxX = -1e30f;
    float minY = 1e30f, maxY = -1e30f;

    // Rotate each corner and find the new bounding box
    for(int i = 0; i < 4; ++i){
        float x = corners[i][0];
        float y = corners[i][1];
        // Rotate
        float rx = x * cosA - y * sinA;
        float ry = x * sinA + y * cosA;
        // Translate back to original center
        rx += cx;
        ry += cy;
        // Update bounding box
        if(rx < minX) minX = rx;
        if(rx > maxX) maxX = rx;
        if(ry < minY) minY = ry;
        if(ry > maxY) maxY = ry;
    }

    camera.xMin = minX;
    camera.xMax = maxX;
    camera.yMin = minY;
    camera.yMax = maxY;
}